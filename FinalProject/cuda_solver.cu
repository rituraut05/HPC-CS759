#include "hip/hip_runtime.h"
#include <cstring>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <string>

#include "helper.cuh"

using namespace std;

// Kernel Fill empty cells for all the N2 boards in the sudoku array with the empty cells index

__global__ void fill_empty_cells(int* sudoku, int total_boards, int* empty_cells, int* num_empty_cells){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    printf("id: %d\n", id);
    while(id < total_boards){
        int* board = sudoku + id * N2;
        int* empty_cells_ptr = empty_cells + id * N2;
        int empty_space_count = 0;
        for(int i = 0; i < N2; i++){
            if(board[i] == 0){
                empty_cells_ptr[empty_space_count] = i;
                empty_space_count++;
            }
        }
        num_empty_cells[id] = empty_space_count;
        id += blockDim.x * gridDim.x;
    }
}


// Iterative Backtracking kernel
__global__ void backtracking(int* sudoku, int total_boards, int* empty_cells, int num_empty_cells, int* solved, int* lock){
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    while(id < total_boards && *solved == 0){
        int* board = sudoku + id * N2;
        int empty_id = 0;

        while(empty_id>=0 && empty_id < num_empty_cells && *solved == 0){
            int empty_space = empty_cells[empty_id];
            int row = empty_space / N;
            int col = empty_space % N;

            int num = board[empty_space] + 1;
            while(num <= N){
                if(check(board, row, col, num)){
                    board[empty_space] = num;
                    empty_id++;
                    break;
                }
                num++;
            }

            if(num > N){
                board[empty_space] = 0;
                empty_id--;
            }
        }
        if(empty_id == num_empty_cells && *solved == 0){

            // make sure that only one thread saves the solution using the atomicXchg
            if(atomicExch(lock, 1) == 0){
                if(*solved == 1){
                    atomicExch(lock, 0);
                    return;
                }
                
                printf("Solved!!!!!!!!!!!!!!!!\n");
                *solved = 1;
                memcpy(sudoku, board, N2*sizeof(int));
                __threadfence();
                atomicExch(lock, 0);
            }
        }
        id += blockDim.x * gridDim.x;
    }
}


// BFS kernel
__global__ void bfs(int* prev_sudoku, int* next_sudoku, int total_boards, int* boards_ptr, int *solved, int max_boards){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    while(id < total_boards){
        int* prev_board = prev_sudoku + id * N2;

        int empty_space;
        if (find_empty(prev_board, empty_space)){
            int row = empty_space / N;
            int col = empty_space % N;

            for(int i = 1; i <= N; i++) {
                if(check(prev_board, row, col, i)){
                    int next_boards_ptr = atomicAdd(boards_ptr, 1);
                    if(next_boards_ptr >= max_boards){
                        *solved = -1;
                        return;
                    }
                    // printf("id: %d, next_boards_ptr: %d, boards_ptr: %d\n", id, next_boards_ptr, *boards_ptr);
                    int* next_board = next_sudoku + next_boards_ptr * N2;
                    memcpy(next_board, prev_board, N2*sizeof(int));
                    // for(int j = 0; j < N2; j++){
                    //     next_board[j] = prev_board[j];
                    // }
                    next_board[empty_space] = i;
                }
            }
        }
        else{
            *solved = 1;
            return;
        }
        id += blockDim.x * gridDim.x;
    }
}
 


int main(int argc, char* argv[]){
    if (argc != 4){
        cout<<"Usage: ./sudoku <threadsPerBlock> <blocksPerGrid> <filename>"<<endl;
        return 0;
    }

    int threadsPerBlock = atoi(argv[1]);
    int blocksPerGrid = atoi(argv[2]);
    string filename = argv[3];

    int* sudoku = new int[N2];
    read_file(filename, sudoku);
    print_sudoku(sudoku);
    cout<<endl;

    int* prev_sudoku = new int[N2];
    int* next_sudoku = new int[N2];

    // 6GB total memory to be used
    // max_boards = 2^28
    // 2^2 * 81 = 324 size of one board
    // 324 * 2^28 = 6GB

    int* max_boards;
    hipMallocManaged(&max_boards, sizeof(int));
    *max_boards= pow(2, 12);
    int tot_size_boards = N2 * (*max_boards);

    hipMallocManaged(&prev_sudoku, tot_size_boards * sizeof(int));
    memset(prev_sudoku, 0, tot_size_boards * sizeof(int));
    memcpy(prev_sudoku, sudoku, N2*sizeof(int));

    hipMallocManaged(&next_sudoku, tot_size_boards * sizeof(int));
    memset(next_sudoku, 0, tot_size_boards * sizeof(int));

    int* boards_ptr;
    hipMallocManaged(&boards_ptr, sizeof(int));
    *boards_ptr = 0;

    int* total_boards;
    hipMallocManaged(&total_boards, sizeof(int));
    *total_boards = 1;

    int* solved;
    hipMallocManaged(&solved, sizeof(int));
    *solved = 0;

    int* empty_cells;
    hipMallocManaged(&empty_cells, N2 * sizeof(int));
    memset(empty_cells, 0, N2 * sizeof(int));

    int* num_empty_cells;
    hipMallocManaged(&num_empty_cells, sizeof(int));
    *num_empty_cells = 0;

    dim3 dimGrid(blocksPerGrid);
    dim3 dimBlock(threadsPerBlock);

    int iter = 0;
    int prev_total_boards = 0;
    
    while(*total_boards < (*max_boards)){
        bfs <<<dimGrid, dimBlock>>> (prev_sudoku, next_sudoku, *total_boards, boards_ptr, solved, *max_boards);
        hipDeviceSynchronize();

        *total_boards = *boards_ptr;
        *boards_ptr = 0;

        if(*solved == 1){
            cout<<"Solved!!!!!!!"<<endl;
            memcpy(sudoku, prev_sudoku, N2*sizeof(int));
            print_sudoku(sudoku);
            write_file("result.txt", sudoku);
            return 0;
        }else if (*solved == -1){
            cout<<"Too many boards..."<<endl;
            cout<<"Starting backtracking..."<<endl;
            *total_boards = prev_total_boards;
            break;
        }

        if (*total_boards == 0){
            cout<<"No solution"<<endl;
            return 0;
        }
                
        cout<<"Iteration:"<<iter<<" Total boards:"<<*total_boards<<endl; 

        int *temp = prev_sudoku;
        prev_sudoku = next_sudoku;
        next_sudoku = temp;
        prev_total_boards = *total_boards;
        iter++;
    }


    hipFree(next_sudoku);
    cout<<"Filling empty cells..."<<endl;
    int count = 0;
    for(int i=0; i<N2; i++){
        if(prev_sudoku[i] == 0){
            
            empty_cells[count] = i;
            count++;
        }
    }
    *num_empty_cells = count;


    cout<<"Backtracking..."<<endl;
    *solved = 0;
    int* lock;
    hipMallocManaged(&lock, sizeof(int));
    *lock = 0;

    backtracking <<<dimGrid, dimBlock>>> (prev_sudoku, *total_boards, empty_cells, *num_empty_cells, solved, lock);
    hipDeviceSynchronize();
    
    if(*solved==1){
        cout<<"Solution: "<<endl;
        memcpy(sudoku, prev_sudoku, N2*sizeof(int));
        print_sudoku(sudoku);
        write_file("result.txt", sudoku);
    }
    else{
        cout<<"No solution"<<endl;
    }
    
    hipFree(prev_sudoku);
    hipFree(boards_ptr);
    hipFree(total_boards);
    hipFree(solved);
    hipFree(empty_cells);
    hipFree(num_empty_cells);

    return 0;
}


__device__ bool check_row(int* &sudoku, int row, int num){
    for(int i = 0; i < N; i++){
        if(sudoku[row*N + i] == num){
            return false;
        }
    }
    return true;
}

__device__ bool check_col(int* &sudoku, int col, int num){
    for(int i = 0; i < N; i++){
        if(sudoku[i*N + col] == num){
            return false;
        }
    }
    return true;
}

__device__ bool check_box(int* &sudoku, int row, int col, int num){

    int row_start = (row/Nr)*Nr;
    int col_start = (col/Nr)*Nr;
    for(int i = 0; i < Nr; i++){
        for(int j = 0; j < Nr; j++){
            if(sudoku[(row_start+i)*N + col_start+j] == num){
                return false;
            }
        }
    }
    return true;
}

__device__ bool check(int* &sudoku, int row, int col, int num){
    return check_row(sudoku, row, num) && check_col(sudoku, col, num) && check_box(sudoku, row, col, num);
}

__device__ bool find_empty(int* &sudoku, int &index){
    for(int i = 0; i < N2; i++){
        if(sudoku[i] == 0){
            index = i;
            return true;
        }
    }
    return false;
}