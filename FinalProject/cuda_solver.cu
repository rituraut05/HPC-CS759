#include "hip/hip_runtime.h"
#include <cstring>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <string>

#include "helper.cuh"

using namespace std;

// Kernel Fill empty cells for all the N2 boards in the sudoku array with the empty cells index

__global__ void fill_empty_cells(int* sudoku, int total_boards, int* empty_cells, int* num_empty_cells){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    printf("id: %d\n", id);
    while(id < total_boards){
        int* board = sudoku + id * N2;
        int* empty_cells_ptr = empty_cells + id * N2;
        int empty_space_count = 0;
        for(int i = 0; i < N2; i++){
            if(board[i] == 0){
                empty_cells_ptr[empty_space_count] = i;
                empty_space_count++;
            }
        }
        num_empty_cells[id] = empty_space_count;
        id += blockDim.x * gridDim.x;
    }
}


// Iterative Backtracking kernel
__global__ void backtracking(int* sudoku, int total_boards, int* empty_cells, int* num_empty_cells, int* solved){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    printf("id: %d, total_boards: %d, solved: %d\n", id, total_boards, *solved);
    while(id < total_boards && *solved == 0){
        int* board = sudoku + id * N2;
        int* empty_cells_ptr = empty_cells + id * N2;
        int empty_space_count = num_empty_cells[id];

        int empty_id = 0;

        while(empty_id>=0 && empty_id < empty_space_count ){
            int empty_space = empty_cells_ptr[empty_id];
            int row = empty_space / N;
            int col = empty_space % N;

            int num = board[empty_space] + 1;
            while(num <= N){
                if(check(board, row, col, num)){
                    board[empty_space] = num;
                    empty_id++;
                    break;
                }
                num++;
            }

            if(num > N){
                board[empty_space] = 0;
                empty_id--;
            }
        }
        if(empty_id == empty_space_count){
            *solved = 1;
            memcpy(sudoku, board, N2*sizeof(int));
            printf("Solved!!!!!!!!!!!!!!!!\n");
            return;
        }

        id += blockDim.x * gridDim.x;
    }
}


// BFS kernel
__global__ void bfs(int* prev_sudoku, int* next_sudoku, int total_boards, int* boards_ptr, int *solved){
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    while(id < total_boards){
        int* prev_board = prev_sudoku + id * N2;

        int empty_space;
        if (find_empty(prev_board, empty_space)){
            int row = empty_space / N;
            int col = empty_space % N;

            for(int i = 1; i <= N; i++){
                if(check(prev_board, row, col, i)){
                    int next_boards_ptr = atomicAdd(boards_ptr, 1);
                    // printf("id: %d, next_boards_ptr: %d, boards_ptr: %d\n", id, next_boards_ptr, *boards_ptr);
                    int* next_board = next_sudoku + next_boards_ptr * N2;
                    memcpy(next_board, prev_board, N2*sizeof(int));
                    next_board[empty_space] = i;
                }
            }
        }
        else{
            *solved = 1;
            return;
        }
        id += blockDim.x * gridDim.x;
    }
} 
 


int main(int argc, char* argv[]){
    if (argc != 4){
        cout<<"Usage: ./sudoku <threadsPerBlock> <blocksPerGrid> <filename>"<<endl;
        return 0;
    }

    int threadsPerBlock = atoi(argv[1]);
    int blocksPerGrid = atoi(argv[2]);
    string filename = argv[3];

    int* sudoku = new int[N2];
    read_file(filename, sudoku);
    print_sudoku(sudoku);
    cout<<endl;

    int* prev_sudoku = new int[N2];
    int* next_sudoku = new int[N2];

    // 6GB total memory to be used
    // max_boards = 2^28
    // 2^2 * 81 = 324 size of one board
    // 324 * 2^28 = 6GB

    int max_boards = pow(2, 20);
    int tot_size_boards = N2 * max_boards;

    hipMallocManaged(&prev_sudoku, tot_size_boards * sizeof(int));
    memset(prev_sudoku, 0, tot_size_boards * sizeof(int));
    memcpy(prev_sudoku, sudoku, N2*sizeof(int));

    hipMallocManaged(&next_sudoku, tot_size_boards * sizeof(int));
    memset(next_sudoku, 0, tot_size_boards * sizeof(int));

    int* boards_ptr;
    hipMallocManaged(&boards_ptr, sizeof(int));
    *boards_ptr = 0;

    int* total_boards;
    hipMallocManaged(&total_boards, sizeof(int));
    *total_boards = 1;

    int* solved;
    hipMallocManaged(&solved, sizeof(int));
    *solved = 0;

    dim3 dimGrid(blocksPerGrid);
    dim3 dimBlock(threadsPerBlock);

    int iter = 0;
    int prev_total_boards = 0;
    
    while(*total_boards < max_boards){
        bfs <<<dimGrid, dimBlock>>> (prev_sudoku, next_sudoku, *total_boards, boards_ptr, solved);
        hipDeviceSynchronize();

        *total_boards = *boards_ptr;
        *boards_ptr = 0;

        if(*solved == 1){
            cout<<"Solved!!!!!!!"<<endl;
            memcpy(sudoku, prev_sudoku, N2*sizeof(int));
            print_sudoku(sudoku);
            write_file("result.txt", sudoku);
            return 0;
        }
        if (*total_boards == 0){
            cout<<"No solution"<<endl;
            return 0;
        }
        else if(*total_boards >= max_boards){
            cout<<"Too many boards..."<<endl;
            cout<<"Starting backtracking..."<<endl;
            *total_boards = prev_total_boards;

            break;
        }        
        cout<<"Iteration:"<<iter<<" Total boards:"<<*total_boards<<endl; 

        int *temp = prev_sudoku;
        prev_sudoku = next_sudoku;
        next_sudoku = temp;
        prev_total_boards = *total_boards;
        iter++;
    }


    cout<<"Filling empty cells..."<<endl;
    int* empty_cells;
    hipMallocManaged(&empty_cells, N2* (*total_boards) * sizeof(int));
    memset(empty_cells, 0, tot_size_boards * sizeof(int));
    cout<<"Empty cells allocated"<<endl;

    int* num_empty_cells;
    hipMallocManaged(&num_empty_cells, (*total_boards) * sizeof(int));
    memset(num_empty_cells, 0, max_boards * sizeof(int));
    cout<<"Num empty cells allocated"<<endl;

    fill_empty_cells <<<dimGrid, dimBlock>>> (prev_sudoku, *total_boards, empty_cells, num_empty_cells);
    hipDeviceSynchronize();


    cout<<"#############################################"<<endl;
    cout<<"Empty cells"<<endl;
    for(int i = 0; i < *total_boards * N2; i++){
        cout<<empty_cells[i]<<" ";
        if((i+1)%N == 0){
            cout<<endl;
        }
        if((i+1)%N2 == 0){
            cout<<endl;
        }
    }

    cout<<"#############################################"<<endl;
    cout<<"Num empty cells"<<endl;
    for(int i = 0; i < *total_boards; i++){
        cout<<num_empty_cells[i]<<" ";
    }

    cout<<"Backtracking..."<<endl;
    backtracking <<<dimGrid, dimBlock>>> (prev_sudoku, *total_boards, empty_cells, num_empty_cells, solved);
    hipDeviceSynchronize();
    
    if(*solved){
        memcpy(sudoku, prev_sudoku, N2*sizeof(int));
        print_sudoku(sudoku);
        write_file("result.txt", sudoku);
    }
    
    
    return 0;
}


__device__ bool check_row(int* &sudoku, int row, int num){
    for(int i = 0; i < N; i++){
        if(sudoku[row*N + i] == num){
            return false;
        }
    }
    return true;
}

__device__ bool check_col(int* &sudoku, int col, int num){
    for(int i = 0; i < N; i++){
        if(sudoku[i*N + col] == num){
            return false;
        }
    }
    return true;
}

__device__ bool check_box(int* &sudoku, int row, int col, int num){

    int row_start = (row/Nr)*Nr;
    int col_start = (col/Nr)*Nr;
    for(int i = 0; i < Nr; i++){
        for(int j = 0; j < Nr; j++){
            if(sudoku[(row_start+i)*N + col_start+j] == num){
                return false;
            }
        }
    }
    return true;
}

__device__ bool check(int* &sudoku, int row, int col, int num){
    return check_row(sudoku, row, num) && check_col(sudoku, col, num) && check_box(sudoku, row, col, num);
}

__device__ bool find_empty(int* &sudoku, int &index){
    for(int i = 0; i < N2; i++){
        if(sudoku[i] == 0){
            index = i;
            return true;
        }
    }
    return false;
}