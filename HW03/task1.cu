#include "hip/hip_runtime.h"
#include<iostream>
using namespace std;
/*
Write a C++ program using CUDA in a file calledtask1.cuwhich computes the factorial of integersfrom 1 to 8, by launching a GPU kernel with 1 block and 8 threads.  Inside the kernel, each threadshould usestd::printfto write outa!=b(followed by a newline), whereais one of the 8 integers,andbis the result ofa!.  (Follow your kernel call with a call tocudaDeviceSynchronize()so thatthe host waits for the kernel to finish printing before returning frommain.)•Compile:nvcc task1.cu -Xcompiler -O3 -Xcompiler -Wall -Xptxas -O3 -std=c++17 -otask1•Run (on Euler, use Slurm sbatch!):./task1•Expected output (showing only 4 out of the 8 lines expected; lines could be out of order):1!=12!=23!=64!=24
*/
__global__ 
void factorial(){
    int tid = threadIdx.x;
    int fact = 1;
    for(int i = 1; i <= tid; i++){
        fact *= i;
    }
    printf("%d!=%d\n", tid, fact);
}
int main(){
    int n = 8;
    factorial<<<1, n>>>;
    hipDeviceSynchronize();
    return 0;
}