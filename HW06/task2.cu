#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>
#include <iostream>
#include "scan.cuh"
/*
Write a test program task2.cu which does the following:
•Create and fill an array of lengthnwith randomfloatnumbers in the range [-1, 1]using managed memory, 
where n is the first command line argument as below.
•Call your scan function to fill another array with the results of the inclusive scan.
•Print the last element of the array containing the output of the inclusive scan operation.
•Print the time taken to run the full scan function in milliseconds using CUDA events.
•Compile:nvcc task2.cu scan.cu -Xcompiler -O3 -Xcompiler -Wall -Xptxas-O3 -std c++17 -o task2
•Run  (where n is a positive integer, n≤threadsperblock*threadsperblock):./task2 n threadsperblock
•Exampled expected output:
1065.3
1.12
*/

using namespace std;

int main(int argc, char *argv[])
{   
    unsigned int n = atoi(argv[1]); //size of array
    unsigned int threads_per_block = atoi(argv[2]); //threads per block
    
    float *input = new float[n]; //input array
    float *output = new float[n]; //output array

    // random number generator
    random_device rd;
    mt19937 gen(rd());
    uniform_real_distribution<> disFloat(-1.0, 1.0);

    //fill array with random numbers
    for (unsigned int i = 0; i < n; i++){
        input[i] = disFloat(gen);
        output[i]=disFloat(gen);
    }

    float *deviceInput, *deviceOutput;
    hipMalloc(&deviceInput, n  * sizeof(float)); //allocate memory on device for input array
    hipMalloc(&deviceOutput, n  * sizeof(float)); //allocate memory on device for output array

    hipMemcpy(deviceInput, input, n*  sizeof(float), hipMemcpyHostToDevice); //copy input array to device
    hipMemcpy(deviceOutput, output, n * sizeof(float), hipMemcpyHostToDevice); //copy output array to device
    
     // Create CUDA events
    hipEvent_t start, stop; 
    hipEventCreate(&start); // create start event
    hipEventCreate(&stop); // create stop event
    hipEventRecord(start); // record start event
    scan(deviceInput,deviceOutput,n,threads_per_block); //call scan function
    hipEventRecord(stop, 0); // record stop event
    hipEventSynchronize(stop); // wait for stop event to complete
    hipDeviceSynchronize(); // wait for device to finish
    hipEventDestroy(start); // destroy start event
    hipEventDestroy(stop); // destroy stop event
    hipMemcpy(output, deviceOutput, n * sizeof(float), hipMemcpyDeviceToHost); //copy output array from device to host
    cout << output[n-1] << endl; //print last element of output array
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop); //calculate elapsed time
    cout << elapsedTime << endl; //print elapsed time
    hipFree(deviceInput); //free device memory
    hipFree(deviceOutput); //free device memory
    delete[] input; //free host memory
    delete[] output; //free host memory
    return 0;
}












