// #include <cstdio>
// #include <iostream>
// #include <fstream>
// #include <sstream>
// #include <math.h>
// #include <random>

// #include "hip/hip_runtime.h"
// #include ""
// #include "matmul.cuh"

// int main(int argc, char* argv[])
// {
//     unsigned int n;
//     unsigned int block_dim;
//     std::istringstream input_1(argv[1]);
//     std::istringstream input_2(argv[2]);
//     if (input_1 >> n && input_2>>block_dim && input_2.eof()) {
//         //create arrays on managed mem
//         int* A, * B, * C;
//         hipMallocManaged(&A, sizeof(int) * n * n);
//         hipMallocManaged(&B, sizeof(int) * n * n);
//         hipMallocManaged(&C, sizeof(int) * n * n);

//         std::random_device entropy_source;
//         std::mt19937 generator(entropy_source());
//         std::uniform_int_distribution<int> dist1(-10,10);
//         for (unsigned int i = 0; i < n*n; i++) {
//             // A[i] = dist1(generator);
//             // B[i] = dist1(generator);
//             A[i] = i;
//             B[i] = 1;
//             C[i] = 0;

//         }

//         // kernel call
//         hipEvent_t startEvent, stopEvent;
//         hipEventCreate(&startEvent);
//         hipEventCreate(&stopEvent);
//         hipEventRecord(startEvent, 0);

//         matmul_1(A, B, C, n, block_dim);

//         hipEventRecord(stopEvent, 0);
//         hipEventSynchronize(stopEvent);
//         float elapsedTime;
//         hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
//         hipEventDestroy(startEvent);
//         hipEventDestroy(stopEvent);

//         std::cout << C[0] << "\n";
//         std::cout << C[n * n - 1] << "\n";
//         std::cout << elapsedTime << "\n";

//         hipFree(A);
//         hipFree(B);
//         hipFree(C);
        
//         //Float
//         float *Af, *Bf, *Cf;
//         hipMallocManaged(&Af, sizeof(float) * n * n);
//         hipMallocManaged(&Bf, sizeof(float) * n * n);
//         hipMallocManaged(&Cf, sizeof(float) * n * n);

//         std::uniform_real_distribution<float> dist2(-1.0, 1.0);
//         for (unsigned int i = 0; i < n * n; i++) {
//             // Af[i] = dist2(generator);
//             // Bf[i] = dist2(generator);
//             Af[i] = i;
//             Bf[i] = 1;
//             Cf[i] = 0;
//         }

//         hipEvent_t startEvent2, stopEvent2;
//         hipEventCreate(&startEvent2);
//         hipEventCreate(&stopEvent2);
//         hipEventRecord(startEvent2, 0);

//         matmul_2(Af, Bf, Cf, n, block_dim);

//         hipEventRecord(stopEvent2, 0);
//         hipEventSynchronize(stopEvent2);
//         float elapsedTime2;
//         hipEventElapsedTime(&elapsedTime2, startEvent2, stopEvent2);
//         hipEventDestroy(startEvent2);
//         hipEventDestroy(stopEvent2);

//         std::cout << Cf[0] << "\n";
//         std::cout << Cf[n * n - 1] << "\n";
//         std::cout << elapsedTime2 << "\n";

//         hipFree(Af);
//         hipFree(Bf);
//         hipFree(Cf);

//         //Double
//         double *Ad, *Bd, *Cd;
//         hipMallocManaged(&Ad, sizeof(double) * n * n);
//         hipMallocManaged(&Bd, sizeof(double) * n * n);
//         hipMallocManaged(&Cd, sizeof(double) * n * n);

//         std::uniform_real_distribution<double> dist3(-1.0, 1.0);
//         for (unsigned int i = 0; i < n * n; i++) {
//             // Ad[i] = dist3(generator);
//             // Bd[i] = dist3(generator);
//             Ad[i] = i;
//             Bd[i] = 1;
//             Cd[i] = 0;

//         }

//         // kernel call
//         hipEvent_t startEvent3, stopEvent3;
//         hipEventCreate(&startEvent3);
//         hipEventCreate(&stopEvent3);
//         hipEventRecord(startEvent3, 0);

//         matmul_3(Ad, Bd, Cd, n, block_dim);

//         hipEventRecord(stopEvent3, 0);
//         hipEventSynchronize(stopEvent3);
//         float elapsedTime3;
//         hipEventElapsedTime(&elapsedTime3, startEvent3, stopEvent3);
//         hipEventDestroy(startEvent3);
//         hipEventDestroy(stopEvent3);

//         std::cout << Cd[0] << "\n";
//         std::cout << Cd[n * n - 1] << "\n";
//         std::cout << elapsedTime3 << "\n";

//         hipFree(Ad);
//         hipFree(Bd);
//         hipFree(Cd);
        
//     }

//     return 0;
// }


#include <cstdio>
#include <iostream>
#include <fstream>
#include <sstream>
#include <math.h>
#include <random>

#include "hip/hip_runtime.h"
#include ""
#include "matmul.cuh"

using namespace std;
int main(int argc, char* argv[])
{
    unsigned int n = atoi(argv[1]);
    unsigned int block_dim = atoi(argv[2]);
        int* hostA = new int[n*n]; // The A matrix
        int* hostB = new int[n*n];// The B matrix
        int* hostC = new int[n*n];// The output C matrix
        int* deviceA;
        int* deviceB;
        int* deviceC;
        hipMalloc(&deviceA, sizeof(int) * n * n);
        hipMalloc(&deviceB, sizeof(int) * n * n);
        hipMalloc(&deviceC, sizeof(int) * n * n);

        random_device rd;
        mt19937 gen(rd());
        uniform_int_distribution<> disInt(-10, 10);

        for (unsigned int i = 0; i < n * n; i++) {
            hostA[i] = disInt(gen);
            hostB[i] = disInt(gen);
            // hostA[i] = 1;
            // hostB[i] = 1;
            hostC[i] = 0;
        }

        hipMemcpy(deviceA, hostA, sizeof(int) * n * n, hipMemcpyHostToDevice);
        hipMemcpy(deviceB, hostB, sizeof(int) * n * n, hipMemcpyHostToDevice);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        matmul_1(deviceA, deviceB, deviceC, n, block_dim);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);

        hipMemcpy(hostC, deviceC, sizeof(int) * n * n, hipMemcpyDeviceToHost);
        cout << hostC[0] << endl;
        cout << hostC[n * n - 1] << endl;
        cout << elapsedTime << endl;

        hipFree(deviceA);
        hipFree(deviceB);
        hipFree(deviceC);

        delete[] hostA;
        delete[] hostB;
        delete[] hostC;

        ///////////////////Float///////////////////////

        float* hostAf = new float[n*n]; // The A matrix
        float* hostBf = new float[n*n];// The B matrix
        float* hostCf = new float[n*n];// The output C matrix
        float* deviceAf;
        float* deviceBf;
        float* deviceCf;
        hipMalloc(&deviceAf, sizeof(float) * n * n);
        hipMalloc(&deviceBf, sizeof(float) * n * n);
        hipMalloc(&deviceCf, sizeof(float) * n * n);


        uniform_real_distribution<float> disFloat(-1.0, 1.0);

        for (unsigned int i = 0; i < n * n; i++) {
            hostAf[i] = disFloat(gen);
            hostBf[i] = disFloat(gen);
            // hostAf[i] = 1;
            // hostBf[i] = 1;
            hostCf[i] = 0;
        }

        hipMemcpy(deviceAf, hostAf, sizeof(float) * n * n, hipMemcpyHostToDevice);
        hipMemcpy(deviceBf, hostBf, sizeof(float) * n * n, hipMemcpyHostToDevice);

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        matmul_2(deviceAf, deviceBf, deviceCf, n, block_dim);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);

        hipMemcpy(hostCf, deviceCf, sizeof(float) * n * n, hipMemcpyDeviceToHost);
        cout << hostCf[0] << endl;
        cout << hostCf[n * n - 1] << endl;
        cout << elapsedTime << endl;

        hipFree(deviceAf);
        hipFree(deviceBf);
        hipFree(deviceCf);

        delete[] hostAf;
        delete[] hostBf;
        delete[] hostCf;


    ///////////////////Double///////////////////////

        double* hostAd = new double[n*n]; // The A matrix
        double* hostBd = new double[n*n];// The B matrix
        double* hostCd = new double[n*n];// The output C matrix
        double* deviceAd;
        double* deviceBd;
        double* deviceCd;
        hipMalloc(&deviceAd, sizeof(double) * n * n);
        hipMalloc(&deviceBd, sizeof(double) * n * n);
        hipMalloc(&deviceCd, sizeof(double) * n * n);


        uniform_real_distribution<double> disDouble(-1.0, 1.0);

        for (unsigned int i = 0; i < n * n; i++) {
            hostAd[i] = disDouble(gen);
            hostBd[i] = disDouble(gen);
            // hostAd[i] = 1;
            // hostBd[i] = 1;
            hostCd[i] = 0;
        }

        hipMemcpy(deviceAd, hostAd, sizeof(double) * n * n, hipMemcpyHostToDevice);
        hipMemcpy(deviceBd, hostBd, sizeof(double) * n * n, hipMemcpyHostToDevice);

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        matmul_3(deviceAd, deviceBd, deviceCd, n, block_dim);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);

        hipMemcpy(hostCd, deviceCd, sizeof(double) * n * n, hipMemcpyDeviceToHost);
        cout << hostCd[0] << endl;
        cout << hostCd[n * n - 1] << endl;
        cout << elapsedTime << endl;

        hipFree(deviceAd);
        hipFree(deviceBd);
        hipFree(deviceCd);

        delete[] hostAd;
        delete[] hostBd;
        delete[] hostCd;
    return 0;
}