#include "stencil.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>

using namespace std;
int main(int argc, char *argv[])
{

    int n = atoi(argv[1]);
    int R = atoi(argv[2]);
    int threads_per_block = atoi(argv[3]);

    random_device rd;
    mt19937 gen(rd());
    uniform_real_distribution<> dis(-1, 1);

    float *image, *output, *mask;
    int mask_size = 2 * R + 1;

    hipMallocManaged((void **)&image, n * sizeof(float));
    hipMallocManaged((void **)&output, n * sizeof(float));
    hipMallocManaged((void **)&mask, mask_size * sizeof(float));

    for (int i = 0; i < n; i++)
    {
        image[i] = static_cast<float>(dis(gen));
    }

    for (int i = 0; i < mask_size; ++i)
    {
        mask[i] = static_cast<float>(dis(gen));
    }

    hipEvent_t start;       // start timer
    hipEvent_t stop;        // stop timer
    hipEventCreate(&start); // create start event
    hipEventCreate(&stop);  // create stop event

    hipEventRecord(start); // start timer

    stencil(image, mask, output, n, R, threads_per_block);

    hipEventRecord(stop);      // stop timer
    hipEventSynchronize(stop); // wait for stop event to complete
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop); // calculate elapsed time
    printf("%f\n", output[n - 1]);
    printf("%f\n", milliseconds); // print elapsed time

    hipFree(image);
    hipFree(output);
    hipFree(mask);
}
