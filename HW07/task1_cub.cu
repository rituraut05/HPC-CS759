#define HIPCUB_STDERR
#include <stdio.h>
#include <stdio.h>
#include <random>
#include <iostream>
#include <cub/util_allocator.cuh>
#include <cub/device/device_reduce.cuh>
#include "cub/util_debug.cuh"
#include <hip/hip_runtime.h>

using namespace std;

hipcub::CachingDeviceAllocator g_allocator(true); // Caching allocator for device memory

int main(int argc, char *argv[])
{
    unsigned int n = atoi(argv[1]); // Number of items to reduce

    // random number generator
    random_device rd;
    mt19937 gen(rd());
    uniform_real_distribution<float> dist(-1.0, 1.0);

    float *hostInput = new float[n]; // input array

    for (unsigned int i = 0; i < n; i++) {
        hostInput[i] = dist(gen); // random number between -1 and 1
    }

    float *deviceInput = NULL; // device input array
    g_allocator.DeviceAllocate((void **)&deviceInput, sizeof(float) * n); // allocate device memory
    hipMemcpy(deviceInput, hostInput, sizeof(float) * n, hipMemcpyHostToDevice); // copy input array to device

    float *deviceOutput = NULL; // device output array
    g_allocator.DeviceAllocate((void **)&deviceOutput, sizeof(float) * 1); // allocate device memory

    // Declare temporary storage
    void *deviceTemp = NULL; 
    size_t deviceTempSize = 0; 

    // Allocate temporary storage
    hipcub::DeviceReduce::Sum(deviceTemp, deviceTempSize, deviceInput, deviceOutput, n);
    g_allocator.DeviceAllocate(&deviceTemp, deviceTempSize);

    // Cuda event to measure time
    hipEvent_t start, stop; // start and stop events
    hipEventCreate(&start); // create start event
    hipEventCreate(&stop); // create stop event
    hipEventRecord(start); // record start event

    // Run reduce
    hipcub::DeviceReduce::Sum(deviceTemp, deviceTempSize, deviceInput, deviceOutput, n);

    hipEventRecord(stop, 0); // record stop event
    hipEventSynchronize(stop); // wait for stop event to complete
    float gpu_sum;
    hipMemcpy(&gpu_sum, deviceOutput, sizeof(float) * 1, hipMemcpyDeviceToHost); // copy output array to host
    printf("%f\n", gpu_sum); // print sum

    // Print time taken
    float milliseconds; // time taken
    hipEventElapsedTime(&milliseconds, start, stop); // calculate time
    printf("%f\n", milliseconds); // print time
    hipEventDestroy(start); // destroy start event
    hipEventDestroy(stop); // destroy stop event

    // Free device memory
    if (deviceInput)
        g_allocator.DeviceFree(deviceInput); 
    if (deviceOutput)
        g_allocator.DeviceFree(deviceOutput);
    if (deviceTemp)
        g_allocator.DeviceFree(deviceTemp);

    return 0;
}